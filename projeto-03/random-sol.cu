#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <cmath>


#define BLOCK_SIZE 32
#define ITERS 100000

__global__ void calc_dist(double *X, double *Y, double *dist, int N) {
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;

    dist[i*N+j] = sqrt(pow((X[i] - X[j]), 2) + pow((Y[i] - Y[j]), 2));
}

__global__ void random_sol(int *solutions, double *costs, double *distances, int N, int nSols) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;


    double cost = 0; 


    for (int k = 0; k < N; k++) {
        solutions[i * N + k] = k;
    }
    
    hiprandState_t st;
    hiprand_init(0, i, 0, &st);
    int idx;

    for (int k = 1; k < N; k++){
        idx = (int) ((N-k) * hiprand_uniform(&st) + k); 

        int tmp = solutions[i * N + k];
        solutions[i * N + k] = solutions[i * N + idx];
        solutions[i * N + idx] = tmp;

        cost += distances[solutions[i * N + k-1] * N + solutions[i * N + k]];
    }

    cost += distances[solutions[i * N] * N + solutions[i * N + N-1]]; 

    costs[i] = cost;

    // for (int k=0; k<i;k++){
    //     for (int j=0;j<i;j++){
    //         if costs[k]<costs[j]{

    //         }
    //     }
    // }

}

int main() {
    int N; std::cin >> N;
    thrust::host_vector<double> host_x(N);
    thrust::host_vector<double> host_y(N);

    double x, y;
    for (int i = 0; i < N; i++) {
        std::cin >> x; 
        std::cin >> y;
        host_x[i] = x;
        host_y[i] = y;
    }

    //copia as coisas para a gpu
    thrust::device_vector<double> dev_x(host_x);
    thrust::device_vector<double> dev_y(host_y);
    thrust::device_vector<double> dev_points_distance(N * N);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(ceil(double N / double threads.x), ceil(double N / double threads.y));

    calc_dist<<<grid,threads>>>(thrust::raw_pointer_cast(dev_x.data()), thrust::raw_pointer_cast(dev_y.data()), thrust::raw_pointer_cast(dev_points_distance.data()), N);


    double nSols = 1024;
    int gpu_threads = 1024;
    
    thrust::device_vector<int> dev_solutions(nSols * N); 
    thrust::device_vector<double> dev_costs(nSols); 


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    
    random_sol<<<ceil((double) nSols/gpu_threads), gpu_threads>>>(thrust::raw_pointer_cast(dev_solutions.data()), thrust::raw_pointer_cast(dev_costs.data()), thrust::raw_pointer_cast(dev_points_distance.data()), N, nSols);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    thrust::device_vector<double>::iterator iter = thrust::min_element(dev_costs.begin(), dev_costs.end());
    int position = iter - dev_costs.begin();
    double min_val = *iter;


    std::cout << min_val;
    std::cout << " 0" << std::endl;

    for (int i = position * N; i < position * N + N; i++) {
        std::cout << dev_solutions[i] << ' ';
    }
    std::cout << std::endl;

    return 0;
}