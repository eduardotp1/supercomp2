#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <cmath>


#define BLOCK_SIZE 32
#define ITERS 100000

__global__ void calc_dist(double *X, double *Y, double *dist, int N) {
    int i = blockIdx.y*blockDim.y+threadIdx.y;
    int j = blockIdx.x*blockDim.x+threadIdx.x;

    if (j >= N){
        return;
    }
    dist[i*N+j] = sqrt(pow((X[i] - X[j]), 2) + pow((Y[i] - Y[j]), 2));
}

__global__ void random_sol(int *solutions, double *costs, double *distances, int N, int nSols) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;

    if (i>nSols){
        return;
    }

    double cost = 0; 


    for (int k = 0; k < N; k++) {
        solutions[i * N + k] = k;
    }
    
    hiprandState_t st;
    hiprand_init(0, i, 0, &st);
    int idx;

    for (int k = 1; k < N; k++){
        idx = (int) ((N-k) * hiprand_uniform(&st) + k); 

        int tmp = solutions[i * N + k];
        solutions[i * N + k] = solutions[i * N + idx];
        solutions[i * N + idx] = tmp;

        cost += distances[solutions[i * N + k-1] * N + solutions[i * N + k]];
    }

    cost += distances[solutions[i * N] * N + solutions[i * N + N-1]]; 

    


    double new_cost = 0;
    for (int k = 1; k < N; k++) {
        for (int j = k + 1; j < N; j++) {
            int temp = solutions[i * N + k];
            solutions[i * N + k] = solutions[i * N + j];
            solutions[i * N + j] = temp;
            double cost = 0;

            for (int k = 1; k < N; k++) {
                cost += distances[solutions[i * N + k-1] * N + solutions[i * N + k]];
                new_cost += distances[solutions[i * N] * N + solutions[i * N + N-1]]; 
                if (new_cost < cost) {
                    cost = new_cost;
                    } 
                else {
                    int temp = solutions[i * N + j];
                    solutions[i * N + j] = solutions[i * N + k];
                    solutions[i * N + k] = temp;
                }
        }
    }
    costs[i] = cost;
    }}

int main() {
    double N;
    std::cin >> N;
    thrust::host_vector<double> host_x(N);
    thrust::host_vector<double> host_y(N);

    double x, y;
    for (int i = 0; i < N; i++) {
        std::cin >> x; 
        std::cin >> y;
        host_x[i] = x;
        host_y[i] = y;
    }

    //copia as coisas para a gpu
    thrust::device_vector<double> dev_x(host_x);
    thrust::device_vector<double> dev_y(host_y);
    thrust::device_vector<double> dev_points_distance(N * N);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(ceil(N / threads.x), ceil(N / threads.y));

    calc_dist<<<grid,threads>>>(thrust::raw_pointer_cast(dev_x.data()), thrust::raw_pointer_cast(dev_y.data()), thrust::raw_pointer_cast(dev_points_distance.data()), N);


    double nSols = 10000;
    int gpu_threads = 1024;
    
    thrust::device_vector<int> dev_solutions(nSols * N); 
    thrust::device_vector<double> dev_costs(nSols); 


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);
    
    random_sol<<<ceil( nSols/gpu_threads), gpu_threads>>>(thrust::raw_pointer_cast(dev_solutions.data()), thrust::raw_pointer_cast(dev_costs.data()), thrust::raw_pointer_cast(dev_points_distance.data()), N, nSols);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 0.0f;
    hipEventElapsedTime(&msecTotal, start, stop);

    thrust::device_vector<double>::iterator iter = thrust::min_element(dev_costs.begin(), dev_costs.end());
    int position = iter - dev_costs.begin();
    double min_val = *iter;


    std::cout << min_val;
    std::cout << " 0" << std::endl;

    for (int i = position * N; i < position * N + N; i++) {
        std::cout << dev_solutions[i] << ' ';
    }
    std::cout << std::endl;

    return 0;
}